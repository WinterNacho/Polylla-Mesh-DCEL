#include "hip/hip_runtime.h"
/* Polygon mesh generator
//POSIBLE BUG: el algoritmo no viaja por todos los halfedges dentro de un poligono, 
    //por lo que pueden haber semillas que no se borren y tener poligonos repetidos de output
*/

#ifndef POLYLLA_HPP
#define POLYLLA_HPP


#include <array>
#include <vector>
#include <string>
#include <iostream>
#include <fstream>
#include <cmath>
#include "kernel.cu"
#include <chrono>
#include <iomanip>

#include <malloc_count-0.7.1/malloc_count.h>
#include <malloc_count-0.7.1/stack_count.h>

#define PARALLEL

#define print_e(eddddge) eddddge<<" ( "<<mesh_input->origin(eddddge)<<" - "<<mesh_input->target(eddddge)<<") "


class GPolylla
{
private:
    typedef std::vector<int> _polygon; 
    typedef std::vector<int> bit_vector; 


    Triangulation *mesh_input; // Halfedge triangulation
    Triangulation *mesh_output;
    std::vector<int> output_seeds; //Seeds of the polygon

    //std::vector<int> triangles; //True if the edge generated a triangle CHANGE!!!!

    bit_vector max_edges; //True if the edge i is a max edge
    bit_vector frontier_edges; //True if the edge i is a frontier edge
    std::vector<int> seed_edges; //Seed edges that generate polygon simple and non-simple




    // Auxiliary array used during the barrier-edge elimination
    std::vector<int> triangle_list;
    
    bit_vector seed_bet_mark;

    // Region options
    bool use_regions = false;

    //Statistics
    int m_polygons = 0; //Number of polygons
    int n_frontier_edges = 0; //Number of frontier edges
    int n_barrier_edge_tips = 0; //Number of barrier edge tips
    int n_polygons_to_repair = 0;
    int n_polygons_added_after_repair = 0;

    // Times Device
    double t_copy_to_device_d = 0;
    double t_label_max_edges_d = 0;
    double t_label_frontier_edges_d = 0;
    double t_label_seed_edges_d = 0;
    double t_label_extra_frontier_edge_d = 0;
    double t_label_seed_scan_d = 0;
    double t_label_seed_compaction_d = 0;
    double t_traversal_and_repair_d = 0;
    double t_traversal_1_d = 0;
    double t_traversal_2_d = 0;
    double t_overwrite_seed_d = 0;
    double t_back_to_host_d = 0;

    // Times Host
    double t_label_max_edges_h = 0;
    double t_label_frontier_edges_h = 0;
    double t_label_seed_edges_h = 0;
    double t_traversal_and_repair_h = 0;
    double t_traversal_h = 0;
    double t_repair_h = 0;

    int n_halfedges;
    int n_triangle;
    int n_vertices;
    int seed_len;

    
public:

    GPolylla() {}; //Default constructor

    //Constructor random data construictor
    GPolylla(int size){
        this->mesh_input = new Triangulation(size);
        std::cout<<"[Polylla] Input mesh generated"<<std::endl;
        mesh_output = new Triangulation(*mesh_input);
        construct_Polylla();
    }

    //Constructor from a OFF file
    GPolylla(std::string off_file, bool use_regions = false){
        this->use_regions = use_regions;
        this->mesh_input = new Triangulation(off_file, use_regions);
        mesh_output = new Triangulation(*mesh_input);
        construct_Polylla();
    }

    //Constructor from a node_file, ele_file and neigh_file
    GPolylla(std::string node_file, std::string ele_file, std::string neigh_file, bool use_regions = false){
        this->use_regions = use_regions;
        this->mesh_input = new Triangulation(node_file, ele_file, neigh_file, use_regions);
        //call copy constructor
        mesh_output = new Triangulation(*mesh_input);
        construct_Polylla();
    }

    ~GPolylla() {
        //triangles.clear(); 
        max_edges.clear(); 
        frontier_edges.clear();
        seed_edges.clear(); 
        seed_bet_mark.clear();
        triangle_list.clear();
        

        delete mesh_input;
        delete mesh_output;
    }

    void construct_Polylla(){

        std::cout<<"[Polylla] Constructing Polylla..."<<std::endl;

        bit_vector_d *max_edges_d;
        halfEdge *halfedges_d, *halfedges_h;
        vertex *vertices_d, *vertices_h;

        max_edges = bit_vector(mesh_input->halfEdges(), 0);
        frontier_edges = bit_vector(mesh_input->halfEdges(), 0);
        seed_bet_mark = bit_vector(this->mesh_input->halfEdges(), 0);


        // copy to device and initialize

        // declare and initialize device arrays
        n_triangle = mesh_input->faces();
        n_halfedges = mesh_input->halfEdges();


        // copy halfedges to device
        halfedges_h = new halfEdge[n_halfedges];
        halfedges_h = mesh_input->HalfEdges.data();

        // copy vertices to device
        n_vertices = mesh_input->vertices();
        vertices_h = new vertex[n_vertices];
        vertices_h = mesh_input->Vertices.data();


        //CUda MAllocs
        hipMalloc(&halfedges_d, n_halfedges*sizeof(halfEdge) );
        hipMalloc(&vertices_d, n_vertices*sizeof(vertex) );
        hipMalloc(&max_edges_d, n_halfedges*sizeof(bit_vector_d) );

        gpuErrchk( hipDeviceSynchronize() ); // clean gpu timers
        auto t_start = std::chrono::high_resolution_clock::now();

        hipMemcpy(halfedges_d, halfedges_h, n_halfedges*sizeof(halfEdge), hipMemcpyHostToDevice );
        hipMemcpy(vertices_d, vertices_h, n_vertices*sizeof(vertex), hipMemcpyHostToDevice );


        hipMemcpy(max_edges_d, max_edges.data(), max_edges.size()*sizeof(bit_vector_d), hipMemcpyHostToDevice );

        gpuErrchk( hipDeviceSynchronize() );
        auto t_end = std::chrono::high_resolution_clock::now();
        t_copy_to_device_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Copy vectors to device in "<<t_copy_to_device_d<<" ms"<<std::endl;
            
        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Label max edges
        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        
        // copy max edges to device
        label_edges_max_d<<<(n_triangle + BSIZE - 1)/BSIZE, BSIZE>>>(max_edges_d, vertices_d, halfedges_d, n_triangle);
        gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_label_max_edges_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labered max edges in "<<t_label_max_edges_d<<" ms"<<std::endl;
       
        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Label frontier edges
            

        bit_vector_d *frontier_edges_d;
        hipMalloc(&frontier_edges_d, sizeof(bit_vector_d)*n_halfedges);
        
        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        
        // Prepare region data for GPU if using regions
        int *triangle_regions_d = nullptr;
        int num_regions = 0;
        if (use_regions && mesh_input->triangle_regions.size() > 0) {
            num_regions = mesh_input->triangle_regions.size();
            hipMalloc(&triangle_regions_d, num_regions * sizeof(int));
            hipMemcpy(triangle_regions_d, mesh_input->triangle_regions.data(), num_regions * sizeof(int), hipMemcpyHostToDevice);
        }
        
        label_phase<<<(n_halfedges + BSIZE - 1)/BSIZE, BSIZE>>>(halfedges_d, max_edges_d, frontier_edges_d, n_halfedges, 
                                                              triangle_regions_d, num_regions, use_regions); 
        hipDeviceSynchronize();
        
        t_end = std::chrono::high_resolution_clock::now();
        t_label_frontier_edges_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labeled frontier edges in "<<t_label_frontier_edges_d<<" ms"<<std::endl;

        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Seed phase


        // GPU SEED PHASE
        half *seed_edges_ad;
        int *seed_edges_d, *seed_edges_bd;
        hipMalloc(&seed_edges_ad, sizeof(half)*n_halfedges);
        hipMemset(seed_edges_ad, 0, sizeof(half)*n_halfedges);
        hipMalloc(&seed_edges_bd, sizeof(int)*n_halfedges);
        hipMalloc(&seed_edges_d, sizeof(int)*n_halfedges);


        //dim3 blockDim(1024,1024,1);
        //dim3 gridDim((n_halfedges+1024*1024-1)/(1024*1024),1,1);

        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Label seed edges 1

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();

        seed_phase_d<<<(n_halfedges + BSIZE - 1)/BSIZE, BSIZE>>>(halfedges_d, max_edges_d, seed_edges_ad, n_halfedges,
                                                             triangle_regions_d, num_regions, use_regions); 
        gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_label_seed_edges_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labeled seed edges in "<<t_label_seed_edges_d<<" ms"<<std::endl;

        //call print_all_halfedges kernel
       //print_all_halfedges<<<(n_halfedges + BSIZE - 1)/BSIZE, BSIZE>>>(halfedges_d, n_halfedges);
        
        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Repair phase

        // print vertex 291
        //std::cout<<"Vertex 291: "<<mesh_input->get_PointX(291)<<" "<<mesh_input->get_PointY(291)<<std::endl;

        //std::cout<<"[GPU] label extra frontier edge kernel..."<<std::endl;

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        
        label_extra_frontier_edge_d<<<(n_vertices + BSIZE - 1)/BSIZE,BSIZE>>>(halfedges_d, frontier_edges_d, vertices_d, seed_edges_ad, n_vertices);
        gpuErrchk( hipDeviceSynchronize() );
        
        t_end = std::chrono::high_resolution_clock::now();
        t_label_extra_frontier_edge_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labeled extra frontier edge in "<<t_label_extra_frontier_edge_d<<" ms"<<std::endl;


        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Travel 

        
        //int *output_seed_d;
        halfEdge *output_HalfEdges_d;
        hipMalloc(&output_HalfEdges_d, sizeof(halfEdge)*n_halfedges);

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        travel_phase_d<<<(n_halfedges + BSIZE - 1)/BSIZE,BSIZE>>>(output_HalfEdges_d, halfedges_d, max_edges_d, frontier_edges_d, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );
        
        t_end = std::chrono::high_resolution_clock::now();
        t_traversal_1_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Traversal phase 1 in "<<t_traversal_1_d<<" ms"<<std::endl;


        //int seed_len = scan(seed_edges_d, seed_edges_ad, n_halfedges); // ESTO SE PUEDE MEJORAR!
        //gpuErrchk( hipDeviceSynchronize() );
        //printf ("-> %i %i %i %i\n", grid.x, grid.y, grid.z, (n_halfedges + BSIZE - 1)/BSIZE);


        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Repair search frontier edge
 

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        search_frontier_edge_d<<<(n_halfedges + BSIZE - 1)/BSIZE,BSIZE>>>(halfedges_d, frontier_edges_d, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_traversal_2_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Traversal phase (search frontier edge) in "<<t_traversal_2_d<<" ms"<<std::endl;


        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Repair overwrite seed

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        
        overwrite_seed_d<<<(n_halfedges + BSIZE - 1)/BSIZE,BSIZE>>>(output_HalfEdges_d, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_overwrite_seed_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Rapair overwrite seed in "<<t_overwrite_seed_d<<" ms"<<std::endl;


        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Label seed edges 2, Scan...

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        scan_parallel_tc_2<int>(seed_edges_bd, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_label_seed_scan_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labeled seed scan in "<<t_label_seed_scan_d<<" ms"<<std::endl;

        hipMemcpy( &seed_len, seed_edges_bd + n_halfedges - 1, sizeof(int), hipMemcpyDeviceToHost );

        //int seed_len = scan(seed_edges_d, seed_edges_ad, n_halfedges); // ESTO SE PUEDE MEJORAR!
        //gpuErrchk( hipDeviceSynchronize() );
        //printf ("-> %i %i %i %i\n", grid.x, grid.y, grid.z, (n_halfedges + BSIZE - 1)/BSIZE);

     


        ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Label seed edges 3, Compaction

        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        compaction_d<<<(n_halfedges + BSIZE - 1)/BSIZE,BSIZE>>>(seed_edges_d, seed_edges_bd, seed_edges_ad, n_halfedges);
        gpuErrchk( hipDeviceSynchronize() );
        //compaction_cub(seed_edges_d, d_num, max_edges_d, seed_edges_ad, n_halfedges);
        //gpuErrchk( hipDeviceSynchronize() );
        //printf("\ndone GPU seed phase....\n\n");//*/

        // copy seed edges to device and print
        //int *seed_edges_ah = new int[seed_len];
        //hipMemcpy(seed_edges_ah, seed_edges_d, seed_len*sizeof(int), hipMemcpyDeviceToHost );
        //printf("\nseed edges: ");
        //for (int i = 0; i < seed_len; i++) {
        //    //if ((int)seed_edges_ah[i] != 0) {
        //        printf("%i ", seed_edges_ah[i]);
        //}
     
        t_end = std::chrono::high_resolution_clock::now();
        t_label_seed_compaction_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Labeled seed compaction in "<<t_label_seed_compaction_d<<" ms"<<std::endl;



        // Back to host
        gpuErrchk( hipDeviceSynchronize() );
        t_start = std::chrono::high_resolution_clock::now();
        bit_vector_d *h_max_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_max_edges, max_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        bit_vector_d *h_frontier_edges = new bit_vector_d[n_halfedges];
        hipMemcpy( h_frontier_edges, frontier_edges_d, n_halfedges*sizeof(bit_vector_d), hipMemcpyDeviceToHost );
        int *h_seed_edges = new int[seed_len];
        hipMemcpy( h_seed_edges, seed_edges_d, seed_len*sizeof(int), hipMemcpyDeviceToHost );
        //int *output_seed_h = new int[seed_len];
        //hipMemcpy(output_seed_h, output_seed_d, sizeof(int)*seed_len, hipMemcpyDeviceToHost);
        //gpuErrchk( hipDeviceSynchronize() );

        t_end = std::chrono::high_resolution_clock::now();
        t_back_to_host_d = std::chrono::duration<double, std::milli>(t_end-t_start).count();
        std::cout<<"[GPU] Back to host in "<<t_back_to_host_d<<" ms"<<std::endl;
         

        // standard output, time not measured
        for (int i = 0; i < n_halfedges; i++)
            max_edges[i] = h_max_edges[i];
        for (int i = 0; i < n_halfedges; i++){
            frontier_edges[i] = h_frontier_edges[i];
            //std::cout<< i <<" "<<h_frontier_edges[i]<<"\n";
        }
        //for (int i = 0; i < seed_len; i++)
        //    seed_edges[i] = h_seed_edges[i];
        //for (int i = 0; i < seed_len; i++)
        //    output_seeds[i] = output_seed_h[i];
        //std::vector<int> aux_seed_edges(h_seed_edges, h_seed_edges + seed_len);
        //std::vector<int> aux_output(output_seed_h, output_seed_h + seed_len);
        //seed_edges = aux_seed_edges;
        //output_seeds = aux_output;
        gpuErrchk( hipDeviceSynchronize() );

        // copy output_halfeget_h to halfedges
        halfEdge *h_halfedges = new halfEdge[n_halfedges];
        hipMemcpy(h_halfedges, output_HalfEdges_d, sizeof(halfEdge)*n_halfedges, hipMemcpyDeviceToHost);
        gpuErrchk( hipDeviceSynchronize() );
        for (int i = 0; i < n_halfedges; i++)
            mesh_output->HalfEdges[i] = h_halfedges[i];


        //print output_seed
        for (int i = 0; i < seed_len; i++){
          //  std::cout<<h_seed_edges[i]<<" ";
            output_seeds.push_back(h_seed_edges[i]);
        }

        this->m_polygons = output_seeds.size();
        std::cout<<"[GPU] Mesh with "<<m_polygons<<" polygons "<<n_frontier_edges/2<<" edges and "<<n_barrier_edge_tips<<" barrier-edge tips."<<std::endl;
        //mesh_input->print_pg(std::to_string(mesh_input->vertices()) + ".pg");    


        // Free GPU memory
        if (triangle_regions_d != nullptr) {
            hipFree(triangle_regions_d);
        }
        
        // hipFree(max_edges_d);
        // hipFree(frontier_edges_d);
        // hipFree(seed_edges_d);
        // hipFree(halfedges_d);
        // hipFree(vertices_d);
        
       // delete h_seed_edges;
       // delete h_frontier_edges;
       // delete h_max_edges;
       // delete output_seed_h;
       // delete halfedges_h;
       // delete vertices_h;
    }


    void print_stats(std::string filename){
        //Time
        /*std::cout<<"Time to generate Triangulation: "<<mesh_input->get_triangulation_generation_time()<<" ms"<<std::endl;
        std::cout<<"Time to copy to device: "<<t_copy_to_device_d<<" ms"<<std::endl;
        std::cout<<"Time to label max edges "<<t_label_max_edges_d<<" ms"<<std::endl;
        std::cout<<"Time to label frontier edges "<<t_label_frontier_edges_d<<" ms"<<std::endl;
        std::cout<<"Time to label seed edges "<<t_label_seed_edges_d<<" ms"<<std::endl;
        std::cout<<"Time to label total"<<t_label_max_edges_d+t_label_frontier_edges_d+t_label_seed_edges_d<<" ms"<<std::endl;
        std::cout<<"Time to traversal and repair "<<t_traversal_and_repair_d<<" ms"<<std::endl;
        std::cout<<"Time to traversal "<<t_traversal_1_d<<" ms"<<std::endl;
        std::cout<<"Time to traversal "<<t_traversal_2_d<<" ms"<<std::endl;
        std::cout<<"Time to repair "<<t_repair_d<<" ms"<<std::endl;
        std::cout<<"Time to back to host: "<<t_back_to_host_d<<" ms"<<std::endl;
        std::cout<<"Time to generate polygonal mesh "<<t_label_max_edges_d + t_label_frontier_edges_d + t_label_seed_edges_d + t_traversal_and_repair_d<<" ms"<<std::endl;//*/

       

        //Memory
        long long m_max_edges =  n_halfedges*sizeof(bit_vector_d);
        long long m_frontier_edge =  sizeof(bit_vector_d)*n_halfedges;
        long long m_seed_edges = sizeof(int)*n_halfedges + sizeof(int)*n_halfedges;
        long long m_seed_bet_mar = sizeof(half)*n_halfedges;
        long long m_triangle_list = sizeof(int)*n_triangle;
        long long m_mesh_input = mesh_input->get_size_vertex_half_edge();
        long long m_mesh_output = mesh_output->get_size_vertex_half_edge();
        long long m_vertices_input = mesh_input->get_size_vertex_struct();
        long long m_vertices_output = mesh_output->get_size_vertex_struct();

        size_t free_byte;
        size_t total_byte;
        hipMemGetInfo(&free_byte, &total_byte);
        size_t m_memory_total = total_byte - free_byte;

        
        std::ofstream out(filename);
        std::cout<<"Printing JSON file as "<<filename<<std::endl;
        out<<"{"<<std::endl;
        out<<"\"parallel\": "<< 1 <<","<<std::endl;
        out<<"\"n_polygons\": "<<m_polygons<<","<<std::endl;
        out<<"\"n_frontier_edges\": "<<n_frontier_edges/2<<","<<std::endl;
        out<<"\"n_barrier_edge_tips\": "<<n_barrier_edge_tips<<","<<std::endl;
        out<<"\"n_half_edges\": "<<mesh_input->halfEdges()<<","<<std::endl;
        out<<"\"n_faces\": "<<mesh_input->faces()<<","<<std::endl;
        out<<"\"n_vertices\": "<<mesh_input->vertices()<<","<<std::endl;
        out<<"\"n_polygons_to_repair\": "<<n_polygons_to_repair<<","<<std::endl;
        out<<"\"n_polygons_added_after_repair\": "<<n_polygons_added_after_repair<<","<<std::endl;
        out<<"\"time_triangulation_generation\": "<<mesh_input->get_triangulation_generation_time()<<","<<std::endl;

        out<<"\"d_time_copy_to_device\": "<<t_copy_to_device_d<<","<<std::endl;
        out<<"\"d_time_to_label_max_edges\": "<<t_label_max_edges_d<<","<<std::endl;
        out<<"\"d_time_to_label_frontier_edges\": "<<t_label_frontier_edges_d<<","<<std::endl;
        out<<"\"d_time_to_label_seed_edges\": "<<t_label_seed_edges_d<<","<<std::endl;
        out<<"\"d_time_to_label_extra_frontier_edge\": "<<t_label_extra_frontier_edge_d<<","<<std::endl;
        out<<"\"d_time_to_label_scan_edges\": "<<t_label_seed_scan_d<<","<<std::endl;
        out<<"\"d_time_to_label_compaction_edges\": "<<t_label_seed_compaction_d<<","<<std::endl;
        out<<"\"d_time_to_label_total\": "<<t_label_max_edges_d+t_label_frontier_edges_d+t_label_seed_edges_d<<","<<std::endl;
        out<<"\"d_time_to_traversal_and_repair\": "<<t_traversal_and_repair_d<<","<<std::endl;
        out<<"\"d_time_to_traversal\": "<<t_traversal_1_d<<","<<std::endl;
        out<<"\"d_time_to_traversal_search_frontier_edge\": "<<t_traversal_2_d<<","<<std::endl;
        out<<"\"d_time_to_back_to_host\": "<<t_back_to_host_d<<","<<std::endl;
        out<<"\"d_time_to_overwrite_seed\": "<<t_overwrite_seed_d<<","<<std::endl;
        out<<"\"d_time_to_generate_polygonal_mesh\": "<<t_label_max_edges_d + t_label_frontier_edges_d + t_label_seed_edges_d + t_traversal_and_repair_d<<","<<std::endl;
        
        out<<"\t\"memory_max_edges\": "<<m_max_edges<<","<<std::endl;
        out<<"\t\"memory_frontier_edge\": "<<m_frontier_edge<<","<<std::endl;
        out<<"\t\"memory_seed_edges\": "<<m_seed_edges<<","<<std::endl;
        out<<"\t\"memory_seed_bet_mar\": "<<m_seed_bet_mar<<","<<std::endl;
        out<<"\t\"memory_triangle_list\": "<<m_triangle_list<<","<<std::endl;
        out<<"\t\"memory_mesh_input\": "<<m_mesh_input<<","<<std::endl;
        out<<"\t\"memory_mesh_output\": "<<m_mesh_output<<","<<std::endl;
        out<<"\t\"memory_vertices_input\": "<<m_vertices_input<<","<<std::endl;
        out<<"\t\"memory_vertices_output\": "<<m_vertices_output<<","<<std::endl;
        out<<"\t\"memory_total\": "<<m_max_edges + m_frontier_edge + m_seed_edges + m_seed_bet_mar + m_triangle_list + m_mesh_input + m_mesh_output + m_vertices_input + m_vertices_output<<","<<std::endl;
        out<<"\t\"memory_total_gpu\": "<<m_memory_total<<std::endl;
        //out<<"\t\"memory_total_cpu\": "<<malloc_count()<<std::endl;
        out<<"}"<<std::endl;
        out.close();
    }



    //Print off file of the polylla mesh
    void print_OFF(std::string filename){
        std::ofstream out(filename);

        std::cout << "Printing OFF file" <<  mesh_input->vertices() << " " << m_polygons << std::endl;

      //  out<<"{ appearance  {+edge +face linewidth 2} LIST\n";
        out<<"OFF"<<std::endl;
        //num_vertices num_polygons 0
        out<<std::setprecision(15)<<mesh_input->vertices()<<" "<<m_polygons<<" 0"<<std::endl;
        //print nodes
        for(std::size_t v = 0; v < mesh_input->vertices(); v++)
            out<<mesh_input->get_PointX(v)<<" "<<mesh_input->get_PointY(v)<<" 0"<<std::endl; 
        //print polygons
        //printf("-------> 1\n");
        int size_poly;
        int e_curr;
        //std::cout<<"aca"<<std::endl;
        for(auto &e_init : output_seeds){
            size_poly = 1;
            e_curr = mesh_output->next(e_init);
            //std::cout<<"poly"<<"e_init"<<e_init<<std::endl;
            while(e_init != e_curr){
              //  std::cout<<"e_init "<<e_init<<" e_curr "<<e_curr<<std::endl;
                size_poly++;
                e_curr = mesh_output->next(e_curr);
            }
            out<<size_poly<<" ";            
            
            out<<mesh_output->origin(e_init)<<" ";
            e_curr = mesh_output->next(e_init);
            while(e_init != e_curr){
                out<<mesh_output->origin(e_curr)<<" ";
                e_curr = mesh_output->next(e_curr);
            }
            
            // Add colors only if using regions
            if (use_regions) {
                // Get region from the original mesh (via first halfedge)
                int region = mesh_input->region_face(mesh_input->index_face(e_init));

                // Generate different RGB colors using prime numbers based on the region
                float r = (region * 73 % 256) / 255.0f;
                float g = (region * 149 % 256) / 255.0f;
                float b = (region * 233 % 256) / 255.0f;

                out << " " << r << " " << g << " " << b << " 1.0";
            }
            out<<std::endl; 
            //count++;
            //printf("-------> 1.1 %d\n", count);
        }
        //printf("-------> 2\n");
      //  out<<"}"<<std::endl;
        out.close();
    }
};

// C wrapper functions for interfacing with C++
extern "C" {
    void* gpolylla_create_default() {
        return new GPolylla();
    }
    
    void* gpolylla_create_int(int size) {
        return new GPolylla(size);
    }
    
    void* gpolylla_create_off(const char* off_file, bool use_regions) {
        return new GPolylla(std::string(off_file), use_regions);
    }
    
    void* gpolylla_create_neigh(const char* node_file, const char* ele_file, const char* neigh_file, bool use_regions) {
        return new GPolylla(std::string(node_file), std::string(ele_file), std::string(neigh_file), use_regions);
    }
    
    void gpolylla_destroy(void* instance) {
        delete static_cast<GPolylla*>(instance);
    }
    
    void gpolylla_print_stats(void* instance, const char* filename) {
        static_cast<GPolylla*>(instance)->print_stats(std::string(filename));
    }
    
    void gpolylla_print_off(void* instance, const char* filename) {
        static_cast<GPolylla*>(instance)->print_OFF(std::string(filename));
    }
}

#endif